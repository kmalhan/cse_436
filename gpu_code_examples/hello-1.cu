
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hellokernel()
{
	printf("Hello World!\n");
}

int main(void)
{
	int num_threads = 10;
	int num_blocks = 10;
	hellokernel<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}
